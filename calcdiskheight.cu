#include "hip/hip_runtime.h"
/** \file glvortens.cu
*/
#define __CUDA 1
#include "fargo.h"
#undef __CUDA
#include <stdarg.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

// BLOCK_X : in azimuth
//#define BLOCK_X DEF_BLOCK_X_GLVORTENS
#define BLOCK_X 32
// BLOCK_Y : in radius
#define BLOCK_Y 4

#define rmed CRadiiStuff[(nr+1)*6 + ig]

__device__  double CRadiiStuff[32768];

__global__ void kernel_adiabatic_diskheight (double *energy,
                                             double *dens,
                                             double *disk_height, 
                                             double  adiabatic_index,
                                             int     nr,
 	                                           int     pitch) {
                               
  int jg = threadIdx.x + blockIdx.x * blockDim.x;
  int ig = threadIdx.y + blockIdx.y * blockDim.y;
  int idg = __mul24(ig, pitch) + jg;

  disk_height[idg] = sqrt((adiabatic_index-1.0)*energy[idg]/dens[idg])*pow(rmed,1.5);
}


void CalcDiskHeight_gpu (PolarGrid *Rho, PolarGrid *Energy, PolarGrid *DiskHeight) {
                    
  int nr = DiskHeight->Nrad;
  int ns = DiskHeight->Nsec;

  //dim3 grid;
  dim3 block = dim3(BLOCK_X, BLOCK_Y);
  dim3 grid ((ns+block.x-1)/block.x, (nr+block.y-1)/block.y);
 
  checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(CRadiiStuff), (void *)RadiiStuff, (size_t)(14*(NRAD+1))*sizeof(double),	0, hipMemcpyHostToDevice));
  
  if (Adiabatic) {
    kernel_adiabatic_diskheight <<<grid, block>>> (Energy->gpu_field,
                                                   Rho->gpu_field,
                                                   DiskHeight->gpu_field, 
                                                   ADIABATICINDEX,
                                                   nr,
                                                   DiskHeight->pitch/sizeof(double));

    hipDeviceSynchronize();
    getLastCudaError ("kernel_adiabatic_calctemp failed");
  }
}
