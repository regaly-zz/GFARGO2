#include "hip/hip_runtime.h"
/** \file calcdustsize.cu
*/
#define __CUDA 1
#include "fargo.h"
#undef __CUDA
#include <stdarg.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

// BLOCK_X : in azimuth
//#define BLOCK_X DEF_BLOCK_X_GLVORTENS
#define BLOCK_X 32
// BLOCK_Y : in radius
#define BLOCK_Y 4

__device__  double CRadiiStuff[32768];

__global__ void kernel_dustsize (double *dust_size_gr,
                                 double *dens,
                                 double *dust_size, 
 	                               int     pitch) {
                               
  int jg = threadIdx.x + blockIdx.x * blockDim.x;
  int ig = threadIdx.y + blockIdx.y * blockDim.y;
  int idg = __mul24(ig, pitch) + jg;

  dust_size[idg] = dust_size_gr[idg]/dens[idg];
}

extern "C"
void CalcDustSize_gpu (PolarGrid *DustSizeGr, PolarGrid *Rho, PolarGrid *DustSize) {
                    
  int nr = DustSize->Nrad;
  int ns = DustSize->Nsec;

  //dim3 grid;
  dim3 block = dim3(BLOCK_X, BLOCK_Y);
  dim3 grid ((ns+block.x-1)/block.x, (nr+block.y-1)/block.y);
 
  checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(CRadiiStuff), (void *)RadiiStuff, (size_t)(14*(NRAD+1))*sizeof(double),	0, hipMemcpyHostToDevice));
  
  kernel_dustsize <<<grid, block>>> (DustSizeGr->gpu_field,
                                     Rho->gpu_field,
                                     DustSize->gpu_field,
                                     DustSize->pitch/sizeof(double));

  hipDeviceSynchronize();
  getLastCudaError ("kernel_dustsize failed");
}
