#include "hip/hip_runtime.h"
/** \file glvortens.cu
*/
#define __CUDA 1
#include "fargo.h"
#undef __CUDA
#include <stdarg.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

// BLOCK_X : in azimuth
//#define BLOCK_X DEF_BLOCK_X_GLVORTENS
#define BLOCK_X 32
// BLOCK_Y : in radius
#define BLOCK_Y 4

__device__  double CRadiiStuff[32768];

__global__ void kernel_synthimg (const double *rho_gr,
                                 const double *rho_sm,
                                 const double *dust_size, 
                                 const double  powindex,
                                       double *brightness,
 	                               const int     pitch) {
                               
  int jg = threadIdx.x + blockIdx.x * blockDim.x;
  int ig = threadIdx.y + blockIdx.y * blockDim.y;
  int idg = __mul24(ig, pitch) + jg;

  temp[idg] = MU/R_SPEC*(adiabatic_index-1.0)*energy[idg]/dens[idg];
}

void CalcSinthImg_gpu (PolarGrid *Rho_gr, PolarGrid *Rho_sm, PolarGrid *DustSize) {
                    
  int nr = Temp->Nrad;
  int ns = Temp->Nsec;

  //dim3 grid;
  dim3 block = dim3(BLOCK_X, BLOCK_Y);
  dim3 grid ((ns+block.x-1)/block.x, (nr+block.y-1)/block.y);
 
  checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(CRadiiStuff), (void *)RadiiStuff, (size_t)(14*(NRAD+1))*sizeof(double),	0, hipMemcpyHostToDevice));
  

  kernel_synthimg <<<grid, block>>> (Rho_gr->gpu_field,
                                     Rho_sm->gpu_field,
                                     DustSize->gpu_field,
                                     DUST_SIZE_DISTR,
                                     Work->gpu_field,
                                     Rho_gr->pitch/sizeof(double));

  hipDeviceSynchronize();
  getLastCudaError ("kernel_synthimg failed");
}
