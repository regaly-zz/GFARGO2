#include "hip/hip_runtime.h"
/** \file glvortens.cu
*/
#define __CUDA 1
#include "fargo.h"
#undef __CUDA
#include <stdarg.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

// BLOCK_X : in azimuth
//#define BLOCK_X DEF_BLOCK_X_GLVORTENS
#define BLOCK_X 32
// BLOCK_Y : in radius
#define BLOCK_Y 4

__device__  double CRadiiStuff[32768];

__global__ void kernel_adiabatic_calctemp (double *energy,
                                           double *dens,
                                           double *temp, 
                                           double  adiabatic_index,
 	                                         int     pitch) {
                               
  int jg = threadIdx.x + blockIdx.x * blockDim.x;
  int ig = threadIdx.y + blockIdx.y * blockDim.y;
  int idg = __mul24(ig, pitch) + jg;

  temp[idg] = MU/R_SPEC*(adiabatic_index-1.0)*energy[idg]/dens[idg];
}

void CalcTemp_gpu (PolarGrid *Rho, PolarGrid *Energy, PolarGrid *Temp) {
                    
  int nr = Temp->Nrad;
  int ns = Temp->Nsec;

  //dim3 grid;
  dim3 block = dim3(BLOCK_X, BLOCK_Y);
  dim3 grid ((ns+block.x-1)/block.x, (nr+block.y-1)/block.y);
 
  checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(CRadiiStuff), (void *)RadiiStuff, (size_t)(14*(NRAD+1))*sizeof(double),	0, hipMemcpyHostToDevice));
  
  if (Adiabatic) {
    kernel_adiabatic_calctemp <<<grid, block>>> (Energy->gpu_field,
                                                 Rho->gpu_field,
                                                 Temp->gpu_field,
                                                 ADIABATICINDEX,
                                                 Temp->pitch/sizeof(double));

    hipDeviceSynchronize();
    getLastCudaError ("kernel_adiabatic_calctemp failed");
  }
}
